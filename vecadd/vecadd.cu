#include "hip/hip_runtime.h"
/*
@modifier : Nilanka Manoj
@compile : nvcc vecadd.cu -o build/vecadd
@run : ./build/vecadd <<n>>
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

double *a, *b;
double *c, *c2;

__global__ void vecAdd(double *A, double *B, double *C, int N)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   C[i] = A[i] + B[i];
}

void vecAdd_h(double *A1, double *B1, double *C1, double N)
{
   for (int i = 0; i < N; i++)
      C1[i] = A1[i] + B1[i];
}

int main(int argc, char **argv)
{
   if (argc == 2)
   {
      printf("=====================round strating==========================\n");
      int n = atoi(argv[1]);
      int nBytes = n * sizeof(double);
      int block_size, block_no;

      a = (double *)malloc(nBytes);
      b = (double *)malloc(nBytes);
      c = (double *)malloc(nBytes);
      c2 = (double *)malloc(nBytes);

      double *a_d, *b_d, *c_d;

      block_size = 768;
      block_no = (int)ceil(n / block_size) + 1;

      for (int i = 0; i < n; i++)
      {
         a[i] = sin(i) * sin(i);
         b[i] = cos(i) * cos(i);
         c[i] = 0;
         c2[i] = 0;
      }

      printf("Allocating device memory on host..\n");
      hipMalloc((void **)&a_d, n * sizeof(double));
      hipMalloc((void **)&b_d, n * sizeof(double));
      hipMalloc((void **)&c_d, n * sizeof(double));

      printf("Copying to device..\n");
      hipMemcpy(a_d, a, n * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(b_d, b, n * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(c_d, c, n * sizeof(double), hipMemcpyHostToDevice);

      printf("Doing GPU Vector add\n");
      clock_t start_d = clock();
      vecAdd<<<block_no, block_size>>>(a_d, b_d, c_d, n);
      hipDeviceSynchronize();
      clock_t end_d = clock();

      printf("Doing CPU Vector add\n");
      clock_t start_h = clock();
      vecAdd_h(a, b, c2, n);
      clock_t end_h = clock();
      double time_d = (double)(end_d - start_d) / CLOCKS_PER_SEC;
      double time_h = (double)(end_h - start_h) / CLOCKS_PER_SEC;

      hipMemcpy(c, c_d, n * sizeof(double), hipMemcpyDeviceToHost);
      printf("Number of elements: %d GPU Time: %f CPU Time: %f\n", n, time_d, time_h);
      hipFree(a_d);
      hipFree(b_d);
      hipFree(c_d);

      int e = memcmp(c, c2, n);
      printf("compaired error : %d\n", e);
   }
   else
   {
      printf("invalid arguments\n");
   }

   return 0;
}